#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// sweep-tt-multistart.c - using VELOCITYBOX and FLOATBOX
// vim: set tabstop=2 softtabstop=2 shiftwidth=2 expandtab :
////////////////////////////////////////////////////////////////////////////////

/********************************************************************************/
/* Given a velocity field v[nx][ny][nz] for a set of points (i,j,k) (where	*/
/* 0 <= i < nx, 0 <= j < ny, 0 <= k < nz) layed out on a grid with delta unit	*/
/* distance, compute the minimum travel time, tt[nx][ny][nz][numstart], for all	*/
/* points to the numstart starting points.  The program is called as follows:	*/
/*										*/
/*	sweep-tt-multistart vfile fsfile startfile       			*/
/*										*/
// vfile is the velocity field file and has the .vbox format.
/*										*/
/* fsfile is the forward star offset file and has the format:			*/
/*										*/
/*	starsize								*/
/*	oi oj ok for every forward star offset (oi,oj,ok)			*/
/*										*/
/* startfile contains starting points and has the format:			*/
/*										*/
/*	numstart								*/
/*	si sj sk for every starting point					*/
/*										*/
/* The program writes to "output.tt" the following:				*/
/*										*/
/*	nx ny nz								*/
/*	tt[i][j][k] for every point (i,j,k) in row-major order			*/
/*										*/
/* for every starting point.							*/
/* (Note, the program currently exits before this is done.)			*/
/********************************************************************************/

#include "iovelocity.h"
#include "timing.h"

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define	FSRADIUSMAX	7	/* maximum radius forward star */
#define	FSMAX		818	/* maximum # of points in a forward star */
#define MODELMAX	250	/* maximum model dimension in X,Y,Z */
#define STARTMAX	4	/* maximum starting points */

#define GRIDX 256
#define GRIDY 256
#define GRIDZ 1
#define BLOCKX 1
#define BLOCKY 1
#define BLOCKZ 64

struct FS {			/* forward start offset */
  int		i, j, k;	/* point coordinates */
  float		d;		/* distance to star center (0,0,0)*/
};

struct MODEL {			/* model point */
  float		v;		/* velocity */
  float		tt[STARTMAX];	/* travel time for starting points */
};

struct START {			/* starting point */
  int		i, j , k;	/* point coordinates */
};

int		changed[STARTMAX];

struct FS	fs[FSMAX];
__constant__ struct FS dc_fs[FSMAX];

struct START	start[STARTMAX];
__constant__ struct START	dc_start[STARTMAX];

struct VELOCITYBOX vbox; // stores JUST velocities
__constant__ struct VELOCITYBOX dc_vbox;

struct FLOATBOX ttboxes[STARTMAX]; // stores JUST travel times, one volume per starting point
__constant__ struct FLOATBOX dc_ttboxes[STARTMAX];


void cudaRun(int, int);
__global__ void
cudaWorker(int, int, int, int, int, int, struct FS *, struct START *, struct VELOCITYBOX *, struct FLOATBOX *,long *);
__device__ int 
sweepXYZ(int, int, int, int, int, int, int, int, struct FS *, float *, float *);

int main(int argc, char* argv[]) {
  int		i, j, k, nx, ny, nz, s;
  int		numradius, starsize, numstart;
  int		fsindex[FSRADIUSMAX];
  float		delta;
  FILE		*fsfile, *ttfile, *startfile;

  const char *velocity_model_file = argv[1];

  /* open velocity model file */
  printf( "Loading velocity model file: %s...", velocity_model_file ); fflush( stdout );
  //if( !vboxloadbinary( &vbox, velocity_model_file ) ) {
  if( !vboxloadtext( &vbox, velocity_model_file ) ) {
    printf( "Cannot open velocity model file: %s\n", velocity_model_file );
    exit(1);
  }
  nx = vbox.box.nx;
  ny = vbox.box.ny;
  nz = vbox.box.nz;
  printf( " done.\n" ); fflush( stdout );
  printf( "Velocity model dimensions: %d x %d x %d\n", nx, ny, nz );

  /* open forward star offset file */
  fsfile = fopen(argv[2],"r");
  if(fsfile == NULL) {
    printf("Cannot open forward star offset file: %s\n", argv[2]);
    exit(1);
  }
  printf("Forward star offset file: %s\n", argv[2]);

  /* open file with starting points */
  startfile = fopen(argv[3],"r");
  if(startfile == NULL) {
    printf("Cannot open starting points file: %s\n", argv[4]);
    exit(1);
  }
  printf("Starting points file: %s\n", argv[3]);

  /* get delta */
  delta = 10.0;
  printf("Delta: %f\n", delta);

  /* read forward star offsets */
  starsize = 0;
  fscanf(fsfile, "%i", &starsize);
  printf("Forward star size: %d\n", starsize);

  for (i=0; i<FSRADIUSMAX; i++) {
    fsindex[i] = 0;
  }
  numradius = 0;
  for (i=0; i<starsize; i++) {
    fscanf(fsfile, "%i %i %i", &fs[i].i, &fs[i].j, &fs[i].k);
    fs[i].d = sqrt(fs[i].i*fs[i].i + fs[i].j*fs[i].j + fs[i].k*fs[i].k);
    if ((numradius+1) < fs[i].d) {
      fsindex[numradius] = i;
      numradius++;
    }
    fs[i].d = delta * fs[i].d; 
  }
  printf("Forward star offsets read\n");
  for (i=0; i<FSRADIUSMAX; i++) {
    printf("numradius: %d, fsindex[%d]: %d\n", numradius, i, fsindex[i]);
  }

  /* read starting points */
  fscanf(startfile, "%i", &numstart);
  // initialize travel times for all starting points
  for( s = 0; s < numstart; s++ ) {
    // prepare travel time volumes
    boxalloc( &ttboxes[s], nx, ny, nz );
    boxsetall( ttboxes[s], INFINITY );

    // set the starting point to have a travel time of 0
    fscanf( startfile, "%i %i %i", &i, &j, &k );
    boxput( ttboxes[s], i, j, k, 0 );
    printf( "starting point %d: %d %d %d\n", s, i, j, k );
    start[s].i = i; start[s].j = j; start[s].k = k;
  }
  printf("Starting points read\n");
  
  hipSetDevice(0);
	cudaRun(numstart, starsize);

  // /* print travel times */
  ttfile = fopen("output.tt","w");
  if(ttfile == NULL) {
    printf("Can not open travel time output file: %s\n", "output.tt");
    exit(1);
  }
  fprintf(ttfile, "%d %d %d\n", nx, ny, nz);
  for (s=0; s<numstart; s++) {
    fprintf(ttfile, "starting point: %d\n", s);
    for (i=0; i<nx; i++) {
      for (j=0; j<ny; j++) {
        for (k=0; k<nz; k++) {
          /* use %g for doubles */

          fprintf(ttfile, "travel time for (%d,%d,%d): %f %d %d %d\n",
            i, j, k, boxget( ttboxes[s], i, j, k ), 0, 0, 0 );
        }
      }
    }
  }
} /* main */

void cudaRun(
    int numstart, 
    int starsize
)
{
  struct FS     *pd_fs;
  struct START  *pd_start;
  struct VELOCITYBOX  *pd_vbox; // stores JUST velocities
  struct FLOATBOX     *pd_ttboxes; // stores JUST travel times, one volume per starting point
  int i, j, nx = vbox.box.nx, ny = vbox.box.ny, nz = vbox.box.nz;
  hipError_t err;
  
  //copy fs to device
  hipMemcpyToSymbol(HIP_SYMBOL(dc_fs), fs, sizeof(fs));
  size_t fssize = sizeof(struct FS)*FSMAX;
  err = hipMalloc( (void**)&pd_fs, fssize );
  if(err != hipSuccess)
    printf("fs malloc error\n");
  err = hipMemcpy( pd_fs, fs, fssize, hipMemcpyHostToDevice );
  if(err != hipSuccess)
    printf("fs copy error: %d\n", (int)fssize);
  printf("1\n");
  
  //copy start points to device
  hipMemcpyToSymbol(HIP_SYMBOL(dc_start), start, sizeof(start));
  size_t startsize = sizeof(struct START)*STARTMAX;
  err = hipMalloc( (void**)&pd_start, startsize );
  if(err != hipSuccess)
    printf("start malloc error\n");
  err = hipMemcpy( pd_start, start, startsize, hipMemcpyHostToDevice );
  if(err != hipSuccess)
    printf("start copy error\n");
  printf("2\n");
  
  //copy velosity box to device
  size_t vboxsize = sizeof(struct VELOCITYBOX);
  size_t flatbytes = (size_t)nx * ny * nz * sizeof(float);
  float *pd_vboxflat;
  err = hipMalloc( (void **)&pd_vbox, vboxsize );
  if(err != hipSuccess)
    printf("vbox malloc error\n");

  err = hipMalloc( (void **)&pd_vboxflat, flatbytes );
  if(err != hipSuccess)
    printf("pd_vboxflat malloc error\n");
  
  struct VELOCITYBOX dummyvbox;
  memcpy( &dummyvbox, &vbox, sizeof(struct VELOCITYBOX) );
  dummyvbox.box.flat = pd_vboxflat;
  err = hipMemcpy( dummyvbox.box.flat, vbox.box.flat, flatbytes, hipMemcpyHostToDevice );
  if(err != hipSuccess)
    printf( "pd_vboxflat copy error\n" );
  err = hipMemcpy( pd_vbox, &dummyvbox, vboxsize, hipMemcpyHostToDevice );
  if(err != hipSuccess)
    printf( "vbox copy error\n" );
	hipMemcpyToSymbol(HIP_SYMBOL(dc_vbox), &dummyvbox, sizeof(dummyvbox));
  printf( "3\n" );
  
  //copy travel time boxes to device
  size_t boxessize = sizeof(struct FLOATBOX)*STARTMAX;
  err = hipMalloc( (void **)&pd_ttboxes, boxessize );
  if(err != hipSuccess)
    printf("boxes malloc error\n");
  
	struct FLOATBOX dummybox[STARTMAX];
  for(i=0; i<STARTMAX; i++){
    float *pd_boxflat;
    err = hipMalloc( (void **)&pd_boxflat, flatbytes );
    if(err != hipSuccess)
      printf("pd_boxflat malloc error\n");
    
    memcpy(dummybox+i, ttboxes+i, sizeof(struct FLOATBOX));
    dummybox[i].flat = pd_boxflat;
    
    err = hipMemcpy( dummybox[i].flat, ttboxes[i].flat, flatbytes, hipMemcpyHostToDevice );
    if(err != hipSuccess)
      printf( "boxflat %d copy error\n", i );
  }
	err = hipMemcpy( pd_ttboxes, dummybox, sizeof(struct FLOATBOX) * STARTMAX, hipMemcpyHostToDevice );
  if(err != hipSuccess)
    printf( "box %d copy error\n", i );
	hipMemcpyToSymbol(HIP_SYMBOL(dc_ttboxes), dummybox, sizeof(dummybox));
	
  printf("4\n");
  
	
  const int tNum = GRIDX * BLOCKX * GRIDY * BLOCKY * GRIDZ * BLOCKZ ;
  //const int blkNum = GRIDX * GRIDY * GRIDZ;
  //const int blkSize = BLOCKX * BLOCKY * BLOCKZ;
  long *pd_anychange, *anychange;
  double sweepTime = 0, dataTransTime = 0;
  
  err = hipMalloc(&pd_anychange, sizeof(long) * tNum);
  if(err != hipSuccess)
    printf( "pd_anychange malloc error\n");
    
  anychange = (long*)malloc(sizeof(long) * tNum);
  printf("5\n");
  
  int nDevices;
  hipGetDeviceCount(&nDevices);
  printf("device: %d\n", nDevices);
  for (int i = 0; i < nDevices; i++) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      printf("Device Number: %d\n", i);
      printf("  Device name: %s\n", prop.name);
      printf("  Memory Clock Rate (KHz): %d\n",
             prop.memoryClockRate);
      printf("  Memory Bus Width (bits): %d\n",
             prop.memoryBusWidth);
      printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
             2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
  
  
  for(i=0; i<numstart; i++){
    long sweepNum = 0, changeSum = 1;
    while (changeSum) {
      sweepTime = 0; dataTransTime = 0; changeSum = 0;
      sweepNum++;
      err = hipMemset(pd_anychange, 0, sizeof(long) * tNum);
      if(err != hipSuccess)
        printf( "pd_anychange memset error\n");
      
			reset_and_start_timer();
      dim3 gridDim(GRIDX,GRIDY,GRIDZ);
      dim3 blockDim(BLOCKX,BLOCKY,BLOCKZ);
      cudaWorker<<<gridDim,blockDim>>>(
        nx, ny, nz, 
        i, 
        0, starsize-1, //Note: change the range to the original starsize only reduce 5ms time.
        dc_fs, 
        dc_start, 
        pd_vbox, 
        pd_ttboxes, 
        pd_anychange
      );
      hipDeviceSynchronize();
      sweepTime = get_elapsed_msec();
      
      if(err != hipSuccess)
        printf("  hipGetLastError() returned %d: %s\n", err, hipGetErrorString(err));
      
      reset_and_start_timer();
      err = hipMemcpy( anychange, pd_anychange, sizeof(long) * tNum, hipMemcpyDeviceToHost );
      if(err != hipSuccess)
        printf( "anychange copy error: %d\n", err);
      dataTransTime = get_elapsed_msec();
      
      for(j = 0; j < tNum; j++){
				changeSum += anychange[j];
      }
      
      printf(" start point: %d, sweep %d: %d changes, sweep %f, data trans %f\n", 
        i, sweepNum, changeSum, sweepTime, dataTransTime);
    }
  }

	printf("6\n");
	
  for(i=0; i<STARTMAX; i++){
    struct FLOATBOX ttboxbuff;
		 err = hipMemcpy( &ttboxbuff, pd_ttboxes+i, sizeof(struct FLOATBOX), hipMemcpyDeviceToHost );
    if(err != hipSuccess)
      printf( "box %d copy error\n", i );
    
		err = hipMemcpy( ttboxes[i].flat, ttboxbuff.flat, flatbytes, hipMemcpyDeviceToHost );
    if(err != hipSuccess)
      printf( "boxflat %d copy error\n", i );
  }
	printf("7\n");
  
  hipFree(pd_fs);
  hipFree(pd_start);
  hipFree(pd_vbox);
  hipFree(pd_vboxflat);
  hipFree(pd_ttboxes);
  hipFree(dummybox[i].flat);
  hipFree(pd_anychange);
  free(anychange);
}

__global__ 
void cudaWorker(
    int d_nx, int d_ny, int d_nz,
    int d_s, 
    int d_starstart, int d_starend,
    struct FS *pd_fs,
    struct START *pd_start,
    struct VELOCITYBOX *pd_vbox,
    struct FLOATBOX *pd_ttboxes,
    long *pd_anychange
)
{
  //int d_blktid = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.z * blockDim.y;
  int d_blkid = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int d_glbtid = d_blkid * (blockDim.x * blockDim.y * blockDim.z)
								+ (threadIdx.z * (blockDim.x * blockDim.y))
								+ (threadIdx.y * blockDim.x) + threadIdx.x;
  //int blkSize = blockDim.x * blockDim.y * blockDim.z;
	
	pd_anychange[d_glbtid] = sweepXYZ(
    d_nx, d_ny, d_nz,
    dc_start[d_s].i, dc_start[d_s].j, dc_start[d_s].k, 
    d_starstart, d_starend, 
    dc_fs, 
    pd_vbox->box.flat,
    pd_ttboxes[d_s].flat
  );
}

__device__ int 
sweepXYZ(
    int nx, int ny, int nz, 
    int startx, int starty, int startz,
    int starstart, int starstop,
    struct FS *fs,
    float *vboxflat,
    float *ttboxflat
) 
{
  int	i, j, k, l, oi, oj, ok;
  float	delay = 0.0, tt = 0.0, tto = 0.0, ttd = 0.0, ttod = 0.0;
  int sx = nz * ny;
  int d_blktid = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.z * blockDim.y;
  __shared__ int change;
  if(d_blktid == 0)
    change = 0;
  __syncthreads();
  
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	k = blockIdx.z * blockDim.z + threadIdx.z;
  
	
	if(i >= nx || j >= ny || k >= nz)
		return 0;
	
	for (l=starstart; l<starstop; l++) {
		/* find point in forward star based on offsets */
		oi = i+fs[l].i; oj = j+fs[l].j; ok = k+fs[l].k;
		/* if (oi,oj,ok) is outside the boundaries, then skip */
		if ((oi < 0) || (oi > nx-1)
				|| (oj < 0) || (oj > ny-1)
				|| (ok < 0) || (ok > nz-1)) {
			continue;
		}
		/* compute delay from (i,j,k) to (oi,oj,ok) with end point average */
    int iIdx = k+nz*j+i*sx; int oIdx = ok+nz*oj+oi*sx;
		delay = dc_fs[l].d * (vboxflat[iIdx] + vboxflat[oIdx]) / 2.0;
		tt = ttboxflat[iIdx];
		tto = ttboxflat[oIdx];

    /* if a shorter travel time through (oi,oj,ok), update (i,j,k) */ 
    if ((delay + tto) < tt) {
      ttboxflat[iIdx] = delay + tto;
      if(change == 0)
        change = 1;
    }
    /* if a shorter travel time through (i,j,k), update (oi,oj,ok) */
    else if ((delay + tt) < tto) {
      ttboxflat[oIdx] = delay + tt;
      if(change == 0)
        change = 1;
    }
  }
  return(change);

} /* end sweepXYZ */ 