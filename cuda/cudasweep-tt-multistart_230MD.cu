#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// sweep-tt-multistart.c - using VELOCITYBOX and FLOATBOX
// vim: set tabstop=2 softtabstop=2 shiftwidth=2 expandtab :
////////////////////////////////////////////////////////////////////////////////

/********************************************************************************/
/* Given a velocity field v[nx][ny][nz] for a set of points (i,j,k) (where	*/
/* 0 <= i < nx, 0 <= j < ny, 0 <= k < nz) layed out on a grid with delta unit	*/
/* distance, compute the minimum travel time, tt[nx][ny][nz][numstart], for all	*/
/* points to the numstart starting points.  The program is called as follows:	*/
/*										*/
/*	sweep-tt-multistart vfile fsfile startfile       			*/
/*										*/
// vfile is the velocity field file and has the .vbox format.
/*										*/
/* fsfile is the forward star offset file and has the format:			*/
/*										*/
/*	starsize								*/
/*	oi oj ok for every forward star offset (oi,oj,ok)			*/
/*										*/
/* startfile contains starting points and has the format:			*/
/*										*/
/*	numstart								*/
/*	si sj sk for every starting point					*/
/*										*/
/* The program writes to "output.tt" the following:				*/
/*										*/
/*	nx ny nz								*/
/*	tt[i][j][k] for every point (i,j,k) in row-major order			*/
/*										*/
/* for every starting point.							*/
/* (Note, the program currently exits before this is done.)			*/
/********************************************************************************/

#include "iovelocity.h"
#include "timing.h"

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define	FSRADIUSMAX	7	/* maximum radius forward star */
#define	FSMAX		818	/* maximum # of points in a forward star */
#define MODELMAX	250	/* maximum model dimension in X,Y,Z */
#define STARTMAX	4	/* maximum starting points */

#define GRIDX 256
#define GRIDY 256
#define GRIDZ 1
#define BLOCKX 1
#define BLOCKY 1
#define BLOCKZ 64
#define DEVNUM 3
const int starSplit[4] = {0, 330, 550, 818};

struct FS {			/* forward start offset */
  int		i, j, k;	/* point coordinates */
  float		d;		/* distance to star center (0,0,0)*/
};

struct MODEL {			/* model point */
  float		v;		/* velocity */
  float		tt[STARTMAX];	/* travel time for starting points */
};

struct START {			/* starting point */
  int		i, j , k;	/* point coordinates */
};

int		changed[STARTMAX];

struct FS	fs[FSMAX];
__constant__ struct FS dc_fs[FSMAX];
struct START	start[STARTMAX];
struct VELOCITYBOX vbox; // stores JUST velocities
struct FLOATBOX ttboxes[STARTMAX]; // stores JUST travel times, one volume per starting point

void cudaRun(int, int);
__global__ 
void cudaWorker(
    int d_nx, int d_ny, int d_nz,
    int d_starstart, int d_starend,
    struct FS *pd_fs,
    float *pd_vboxflat,
    float *pd_ttboxflat,
    long *pd_anychange
);
__device__ int 
sweepXYZ(
    int nx, int ny, int nz, 
    int starstart, int starstop,
    struct FS *fs,
    float *vboxflat,
    float *ttboxflat
);

int main(int argc, char* argv[]) {
  int		i, j, k, nx, ny, nz, s;
  int		numradius, starsize, numstart;
  int		fsindex[FSRADIUSMAX];
  float		delta;
  FILE		*fsfile, *ttfile, *startfile;

  const char *velocity_model_file = argv[1];

  /* open velocity model file */
  printf( "Loading velocity model file: %s...", velocity_model_file ); fflush( stdout );
  //if( !vboxloadbinary( &vbox, velocity_model_file ) ) {
  if( !vboxloadtext( &vbox, velocity_model_file ) ) {
    printf( "Cannot open velocity model file: %s\n", velocity_model_file );
    exit(1);
  }
  nx = vbox.box.nx;
  ny = vbox.box.ny;
  nz = vbox.box.nz;
  printf( " done.\n" ); fflush( stdout );
  printf( "Velocity model dimensions: %d x %d x %d\n", nx, ny, nz );

  /* open forward star offset file */
  fsfile = fopen(argv[2],"r");
  if(fsfile == NULL) {
    printf("Cannot open forward star offset file: %s\n", argv[2]);
    exit(1);
  }
  printf("Forward star offset file: %s\n", argv[2]);

  /* open file with starting points */
  startfile = fopen(argv[3],"r");
  if(startfile == NULL) {
    printf("Cannot open starting points file: %s\n", argv[4]);
    exit(1);
  }
  printf("Starting points file: %s\n", argv[3]);

  /* get delta */
  delta = 10.0;
  printf("Delta: %f\n", delta);

  /* read forward star offsets */
  starsize = 0;
  fscanf(fsfile, "%i", &starsize);
  printf("Forward star size: %d\n", starsize);

  for (i=0; i<FSRADIUSMAX; i++) {
    fsindex[i] = 0;
  }
  numradius = 0;
  for (i=0; i<starsize; i++) {
    fscanf(fsfile, "%i %i %i", &fs[i].i, &fs[i].j, &fs[i].k);
    fs[i].d = sqrt(fs[i].i*fs[i].i + fs[i].j*fs[i].j + fs[i].k*fs[i].k);
    if ((numradius+1) < fs[i].d) {
      fsindex[numradius] = i;
      numradius++;
    }
    fs[i].d = delta * fs[i].d; 
  }
  printf("Forward star offsets read\n");
  for (i=0; i<FSRADIUSMAX; i++) {
    printf("numradius: %d, fsindex[%d]: %d\n", numradius, i, fsindex[i]);
  }

  /* read starting points */
  fscanf(startfile, "%i", &numstart);
  // initialize travel times for all starting points
  for( s = 0; s < numstart; s++ ) {
    // prepare travel time volumes
    boxalloc( &ttboxes[s], nx, ny, nz );
    boxsetall( ttboxes[s], INFINITY );

    // set the starting point to have a travel time of 0
    fscanf( startfile, "%i %i %i", &i, &j, &k );
    boxput( ttboxes[s], i, j, k, 0 );
    printf( "starting point %d: %d %d %d\n", s, i, j, k );
    start[s].i = i; start[s].j = j; start[s].k = k;
  }
  printf("Starting points read\n");
  
  int nDevices;
  hipGetDeviceCount(&nDevices);
  printf("device: %d\n", nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
  
	cudaRun(numstart, starsize);

  /* print travel times */
  ttfile = fopen("output.tt","w");
  if(ttfile == NULL) {
    printf("Can not open travel time output file: %s\n", "output.tt");
    exit(1);
  }
  fprintf(ttfile, "%d %d %d\n", nx, ny, nz);
  for (s=0; s<numstart; s++) {
    fprintf(ttfile, "starting point: %d\n", s);
    for (i=0; i<nx; i++) {
      for (j=0; j<ny; j++) {
        for (k=0; k<nz; k++) {
          /* use %g for doubles */

          fprintf(ttfile, "travel time for (%d,%d,%d): %f %d %d %d\n",
            i, j, k, boxget( ttboxes[s], i, j, k ), 0, 0, 0 );
        }
      }
    }
  }
} /* main */

void cudaRun(
    int numstart, 
    int starsize
)
{
  //constants
  const int blkNum = GRIDX * GRIDY * GRIDZ;
  const int blkSize = BLOCKX * BLOCKY * BLOCKZ;
  const int tNum = blkNum * blkSize;

  //host variables
  long anychange[DEVNUM][tNum];
  int i, j, nx = vbox.box.nx, ny = vbox.box.ny, nz = vbox.box.nz, devIdx = 0, devNum = DEVNUM;
  int nCells = nx * ny * nz;
  size_t flatbytes = (size_t)nCells * sizeof(float);
  float ttflatbuffs[DEVNUM][nCells];
  hipError_t err;
  
  //Cuda variables
  dim3 gridDim(GRIDX,GRIDY,GRIDZ);
  dim3 blockDim(BLOCKX,BLOCKY,BLOCKZ);
  float *pd_vboxflat[DEVNUM];
  float *ppd_ttboxflat[DEVNUM][STARTMAX];
  long *pd_anychange[DEVNUM];
  hipStream_t streams[DEVNUM];
  
  //allocate device memory
  for(devIdx = 0; devIdx < devNum; devIdx++){
    hipSetDevice(devIdx);
    err = hipMalloc((void **)&pd_vboxflat[devIdx], flatbytes);
    if(err != hipSuccess)
      printf("pd_vboxflat malloc error\n");
    err = hipMalloc((void **)&pd_anychange[devIdx], sizeof(long) * tNum);
    if(err != hipSuccess)
      printf( "pd_anychange malloc error\n");
    for(i=0; i<STARTMAX; i++){
      err = hipMalloc((void **)&ppd_ttboxflat[devIdx][i], flatbytes);
      if(err != hipSuccess)
        printf("ppd_ttboxflat malloc error\n");
    }
  }

  //lock up host memory for async transfer
  hipHostRegister(fs, sizeof(fs), hipHostRegisterDefault);
  hipHostRegister(start, sizeof(start), hipHostRegisterDefault);
  hipHostRegister(vbox.box.flat, flatbytes, hipHostRegisterDefault);
  for(i=0; i<STARTMAX; i++)
    hipHostRegister(ttboxes[i].flat, flatbytes, hipHostRegisterDefault);
  
  //async copy memory from host to device
  for(devIdx = 0; devIdx < devNum; devIdx++){
    hipSetDevice(devIdx);
    hipStreamCreate(&streams[devIdx]);
    
    //copy fs to device
    err = hipMemcpyToSymbolAsync(HIP_SYMBOL(dc_fs), fs, sizeof(fs), 0, hipMemcpyHostToDevice, streams[devIdx]);
    if(err != hipSuccess)
      printf("dc_fs copy error\n");
    printf("1\n");
    
    //copy velosity box to device
    err = hipMemcpyAsync(pd_vboxflat[devIdx], vbox.box.flat, flatbytes, hipMemcpyHostToDevice, streams[devIdx]);
    if(err != hipSuccess)
      printf( "pd_vboxflat copy error\n" );
    printf( "2\n" );
    
    //copy travel time boxes to device
    for(i=0; i<STARTMAX; i++){
      err = hipMemcpyAsync(ppd_ttboxflat[devIdx][i], ttboxes[i].flat, flatbytes, hipMemcpyHostToDevice, streams[devIdx]);
      if(err != hipSuccess)
        printf( "ppd_ttboxflat %d copy error\n", i );
    }
    printf("3\n");
  }
  hipStreamSynchronize(0);
  
  //run algorithm
  double tSweep = 0.0, tChangeTrans = 0.0, tSum = 0.0, tMerge = 0.0, tBoxTrans = 0.0, tTotal = 0.0;
  for(i=0; i<numstart; i++){
    long sweepNum = 0, changeSum = 1;
    while (changeSum) {//run until no changes
      changeSum = 0;
      sweepNum++;
      
      //run splited forward stars on different devices
      reset_and_start_timer();
      for(devIdx=0; devIdx<devNum; devIdx++){
        hipSetDevice(devIdx);
        err = hipMemset(pd_anychange[devIdx], 0, sizeof(long) * tNum);
        if(err != hipSuccess)
          printf( "pd_anychange memset error\n");
        
        cudaWorker<<<gridDim,blockDim>>>(
          nx, ny, nz, 
          starSplit[devIdx], starSplit[devIdx+1]-1, //Note: change the range to the original starsize only reduce 5ms time.
          dc_fs, 
          pd_vboxflat[devIdx], 
          ppd_ttboxflat[devIdx][i],
          pd_anychange[devIdx]
        );
      }
      hipStreamSynchronize(0); //sync all devices
      tSweep = get_elapsed_msec();
      
      if(err != hipSuccess) //check error
        printf("  hipGetLastError() returned %d: %s\n", err, hipGetErrorString(err));
      
      //pull back and check changes
      reset_and_start_timer();
      for(devIdx=0; devIdx<devNum; devIdx++){
        hipSetDevice(devIdx);
        err = hipMemcpyAsync(anychange[devIdx], pd_anychange[devIdx], sizeof(long) * tNum, hipMemcpyDeviceToHost, streams[devIdx]);
        if(err != hipSuccess)
          printf("anychange copy error: %d\n", err);
      }
      hipStreamSynchronize(0); //sync all devices
      tChangeTrans = get_elapsed_msec();
      
      reset_and_start_timer();
      for(devIdx=0; devIdx<devNum; devIdx++)
        for(j = 0; j < tNum; j++)
          changeSum += anychange[devIdx][j];
      tSum = get_elapsed_msec();
      
      //copy travel time back from all devices and merge them and then put them back
      reset_and_start_timer();
      for(devIdx = 0; devIdx < devNum; devIdx++){
        hipSetDevice(devIdx);
        err = hipMemcpyAsync(ttflatbuffs[devIdx], ppd_ttboxflat[devIdx][i], flatbytes, hipMemcpyDeviceToHost, streams[devIdx]);
        if(err != hipSuccess)
          printf( "ppd_ttboxflat %d copy error\n", i );
      }
      hipStreamSynchronize(0); //sync all devices
      tBoxTrans = get_elapsed_msec();
      reset_and_start_timer();
      float res;
      for(j = 0; j < nCells; j++){
        res = INFINITY;
        for(devIdx = 0; devIdx < devNum; devIdx++)
          res = fminf(res, ttflatbuffs[devIdx][j]);
        ttboxes[i].flat[j] = res;
      }
      tMerge = get_elapsed_msec();
      for(devIdx = 0; devIdx < devNum; devIdx++){
        hipSetDevice(devIdx);
        err = hipMemcpyAsync(ppd_ttboxflat[devIdx][i], ttboxes[i].flat, flatbytes, hipMemcpyHostToDevice, streams[devIdx]);
        if(err != hipSuccess)
          printf( "ppd_ttboxflat %d copy error\n", i );
      }
      hipStreamSynchronize(0); //sync all devices
      
      tTotal = tSweep + tChangeTrans + tSum + tBoxTrans + tMerge;
      printf(" start point: %d, sweep %d: %d changes, sweep %g, change trans %g\n\
sum %g, box trans %g, merg %g, total %g\n", 
        i, sweepNum, changeSum, tSweep, tChangeTrans, tSum, tBoxTrans, tMerge, tTotal);
    }
    memcpy( ttboxes[i].flat, ttflatbuffs[0], flatbytes);
  }
	printf("6\n");
  
  for(devIdx=0; devIdx<devNum; devIdx++){
    hipFree(pd_vboxflat[devIdx]);
    hipFree(pd_anychange[devIdx]);
    for(i=0; i<STARTMAX; i++)
      err = hipFree(ppd_ttboxflat[devIdx][i]);
    hipStreamDestroy(streams[devIdx]);
  }
  
}

__global__ 
void cudaWorker(
    int d_nx, int d_ny, int d_nz,
    int d_starstart, int d_starend,
    struct FS *pd_fs,
    float *pd_vboxflat,
    float *pd_ttboxflat,
    long *pd_anychange
)
{
  //int d_blktid = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.z * blockDim.y;
  int d_blkid = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
	int d_glbtid = d_blkid * (blockDim.x * blockDim.y * blockDim.z)
								+ (threadIdx.z * (blockDim.x * blockDim.y))
								+ (threadIdx.y * blockDim.x) + threadIdx.x;
  //int blkSize = blockDim.x * blockDim.y * blockDim.z;
	
	pd_anychange[d_glbtid] = sweepXYZ(
    d_nx, d_ny, d_nz,
    d_starstart, d_starend, 
    dc_fs, 
    pd_vboxflat,
    pd_ttboxflat
  );
}

__device__ int 
sweepXYZ(
    int nx, int ny, int nz, 
    int starstart, int starstop,
    struct FS *fs,
    float *vboxflat,
    float *ttboxflat
) 
{
  int	i, j, k, l, oi, oj, ok, iIdx, oIdx;
  float	delay = 0.0, tt = 0.0, tto = 0.0, ttd = 0.0, ttod = 0.0;
  int sx = nz * ny;
  int d_blktid = threadIdx.z + threadIdx.y * blockDim.z + threadIdx.x * blockDim.z * blockDim.y;
  __shared__ int change;
  if(d_blktid == 0)
    change = 0;
  __syncthreads();
  
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	k = blockIdx.z * blockDim.z + threadIdx.z;
  
	
	if(i >= nx || j >= ny || k >= nz)
		return 0;
	
	for (l=starstart; l<starstop; l++) {
		/* find point in forward star based on offsets */
		oi = i+fs[l].i; oj = j+fs[l].j; ok = k+fs[l].k;
		/* if (oi,oj,ok) is outside the boundaries, then skip */
		if ((oi < 0) || (oi > nx-1)
				|| (oj < 0) || (oj > ny-1)
				|| (ok < 0) || (ok > nz-1)) {
			continue;
		}
		//pre-compute all the needed values
    iIdx = k+nz*j+i*sx; oIdx = ok+nz*oj+oi*sx;
		delay = fs[l].d * (vboxflat[iIdx] + vboxflat[oIdx]) / 2.0;
		tt = ttboxflat[iIdx];
		tto = ttboxflat[oIdx];
    ttd = tt + delay;
    ttod = tto + delay;
    
    //if the difference between two values is greater than delay
    //do value switches using pre-calculated values.
    if(fabs(tt-tto) > delay){
      ttboxflat[iIdx] = fminf(tt, ttod);
      ttboxflat[oIdx] = fminf(tto, ttd);
      if(change == 0 && (ttod < tt || ttd < tto))
        change = 1;
    }
  }
  return(change);

} /* end sweepXYZ */ 